
#include <hip/hip_runtime.h>
extern "C" __global__ void Ckernel(float *a, float b)
{
  a[threadIdx.x] = b;
}

extern "C" __device__ float Cdevicefun(float a)
{
  return 2*a;
}
