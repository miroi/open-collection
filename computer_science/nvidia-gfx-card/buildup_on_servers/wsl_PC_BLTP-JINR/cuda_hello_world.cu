/* 

https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/   

*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
 //   std::cout << "Hello World from GPU!\n"  << std::endl;
}

int main() {
    cuda_hello<<<1,1>>>(); 
    printf("Hello World from printf !\n");
    return 0;
}
